#include "cudatest.h"

int main() {
	/*
	 *
	 * MATRIZEN ERZEUGEN
	 *
	 */

	Matrix a = m_new(10,10);
	m_filllimit(a,-3,3);
	m_stat(a);

	Matrix b = m_new(10,10);
	m_filllimit(b,-3,3);
	m_stat(b);

	m_print(m_multiply(a,b));
	m_print(m_multiply(a,m_multiply(a,b)));

	
	/*
	 *
	 * CUDA
	 *
	 */

	printf("CUDAing\n");

	hipSetDevice(0);

	Matrix da, db, dc, c;
	size_t size;

	c = m_new(a.rows, b.cols);

	da.rows = a.rows;
	da.cols = a.cols;

	db.rows = b.rows;
	db.cols = b.cols;

	dc.rows = a.rows;
	dc.cols = b.cols;

	size = da.rows * da.cols * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void**) &da.elements, size));
	CUDA_SAFE_CALL(hipMemcpy(da.elements,a.elements, size, hipMemcpyHostToDevice));

	size = db.rows * db.cols * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void**) &db.elements, size));
	CUDA_SAFE_CALL(hipMemcpy(db.elements,b.elements, size, hipMemcpyHostToDevice));

	size = dc.rows * dc.cols * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void**) &dc.elements, size));

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(b.cols / dimBlock.x, a.rows / dimBlock.y);

	MatrixMultiplyKernel<<<dimGrid, dimBlock>>>(da, db, dc);
	CUT_CHECK_ERROR("Kernel execution failed\n");

	size = dc.rows * dc.cols * sizeof(float);
	hipMemcpy(c.elements, dc.elements, size, hipMemcpyDeviceToHost);

	m_print(c);

	MatrixMultiplyKernel<<<dimGrid, dimBlock>>>(da, dc, dc);
	CUT_CHECK_ERROR("Kernel execution failed\n");

	size = dc.rows * dc.cols * sizeof(float);
	hipMemcpy(c.elements, dc.elements, size, hipMemcpyDeviceToHost);

	m_print(c);

	return 0;
}
