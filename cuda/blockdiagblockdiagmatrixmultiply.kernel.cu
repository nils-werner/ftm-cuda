#include "hip/hip_runtime.h"
#include "blockdiagblockdiagmatrixmultiply.kernel.h"

__global__ void BlockDiagBlockDiagMatrixMultiplyKernel(Matrix A, Matrix B, Matrix C, int blocksize) {
	float sum = 0;
	int from, to;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	from = blocksize * (i / blocksize);
	to = from + blocksize;

	for (int k = from; k < to; ++k) {
		sum += A.elements[from * A.cols + k] * B.elements[k * B.cols + j];
	}
	C.elements[from * C.cols + j] = sum;
}
