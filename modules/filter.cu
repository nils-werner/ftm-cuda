#include "hip/hip_runtime.h"
#include "iirfilter.h"

String string;
Synthesizer synth;
Matrix MatrixC, MatrixA, state;
Matrix MatrixAp, MatrixCA;

/**
 * Wrapper for the methods required in the filter, just calls them in the correct order
 *
 * @param int length
 * @return int 0
 */

int filter(float length, int samples, int blocksize, int filters) {
	initializeCoefficients(length, blocksize, samples, filters);

	float * output = (float *) malloc(sizeof(float) * synth.samples);
	createMatrices();
	createBlockprocessingMatrices();

#if DEBUG == 2
	printf("MatrixA");
	m_print(MatrixA);
	printf("MatrixC");
	m_print(MatrixC);
#endif

	generateSignalGPU(output, string, synth);
	writeFile("filter.wav", output, synth.samples, synth.T);
	return 0;
}











/**
 * Initializes coefficients for a standard Nylon-b-String. The values are, with exception for
 * length (l) are hard-coded according to the values in \cite{rabenstein03}.
 *
 * @param float length
 * @return void
 */

void initializeCoefficients(float length, int blocksize, int samples, int filters) {
	// Saiten-Koeffizienten
	string.l = length;
	string.Ts = 60.97;
	string.rho = 1140;
	string.A = 0.5188e-6;
	string.E = 5.4e9;
	string.I = 0.171e-12;
	string.d1 = 8e-5;
	string.d3 = -1.4e-5;

	// Abtastpunkt
	string.xa = 0.1;

	// Abtastrate und Samplelänge
	synth.T = 44100;
	synth.seconds = 10;
	synth.samples = samples;
	synth.filters = filters;
	synth.blocksize = blocksize;

	assert(synth.samples % synth.blocksize == 0);
}















/**
 * Creates the required matrices by calculating the required number of poles using the equations
 * to be found in \cite{rabenstein03}. The matrices generated are not yet in blockprocessing form.
 *
 * @param void
 * @return void
 */

void createMatrices() {
	int i, mu;
	double gamma, sigma;
	double omega;
	double a, b, c1, c0;

	m_new(&MatrixC, 1, 2 * synth.filters);
	m_new(&MatrixA, 2 * synth.filters, 2 * synth.filters); //BLOCKMATRIX
	m_new(&state, 2 * synth.filters, 1);

	for(i = 0; i < synth.filters; i++) {
		mu = i+1;
		gamma = mu * ( M_PI / string.l );
		sigma = (1 / (2 * string.rho * string.A) ) * (string.d3 * pow(gamma,2) - string.d1);
		omega = sqrt(
				  (
					(
						(string.E * string.I)/(string.rho * string.A)
					      - pow(string.d3, 2)/pow(2 * string.rho * string.A, 2)
					) * pow(gamma, 4)
				  )
				+ (	(
						(string.Ts)/(string.rho * string.A) 
					      + (string.d1 + string.d3)/(2*pow(string.rho*string.A,2))
					) * pow(gamma, 2) )
				+ (
					pow((string.d1)/(2 * string.rho * string.A), 2)
				  )
			);

		a = sin(mu * M_PI * string.xa / string.l);

		b = synth.T * sin(omega * 1 / synth.T) / (omega * 1 / synth.T);
		c1 = -2 * exp(sigma * 1 / synth.T) * cos(omega * 1 / synth.T);
		c0 = exp( 2 * sigma * 1 / synth.T);

#if DEBUG == 1
		printf("%d %d sigma %f\n", i, mu, sigma);
		printf("      omega %f\n", omega);
#endif

		m_set(MatrixC, 0, 2*i  , 0);
		m_set(MatrixC, 0, 2*i+1, a);

		m_set(MatrixA, 2*i  , 2*i  , 0);
		m_set(MatrixA, 2*i  , 2*i+1, -c0);
		m_set(MatrixA, 2*i+1, 2*i  , 1);
		m_set(MatrixA, 2*i+1, 2*i+1, -c1);

		m_set(state, 2*i  , 0, 0);
		m_set(state ,2*i+1, 0, 1);

	}
}
















/**
 * Generates matrices suitable for blockprocessing
 *
 * @param int blocksize
 * @return void
 */ 

void createBlockprocessingMatrices() {
	int i, j;
	Matrix MatrixCA_line, MatrixAp_tmp;
	Matrix *pointer_MatrixAp, *pointer_MatrixAp_tmp;

	pointer_MatrixAp = &MatrixAp;
	pointer_MatrixAp_tmp = &MatrixAp_tmp;

	m_new(&MatrixCA, synth.blocksize, MatrixA.cols);
	m_new(&MatrixAp, MatrixA.rows, MatrixA.cols); // BLOCKDIAGMATRIX
	m_identity(MatrixAp);

	m_prepare_multiply(MatrixC, MatrixAp, &MatrixCA_line);
	m_prepare_multiply(MatrixAp, MatrixA, &MatrixAp_tmp);

	for(i = 1; i <= synth.blocksize; i++) {
		m_multiply(MatrixC, *pointer_MatrixAp, &MatrixCA_line);
		for(j = 0; j < MatrixCA_line.cols; j++) {
			m_set(MatrixCA, i-1, j, m_get(MatrixCA_line, 0, j));
		}
		m_multiplyblockdiag(*pointer_MatrixAp, MatrixA, pointer_MatrixAp_tmp, 2);

		m_swap(&pointer_MatrixAp_tmp, &pointer_MatrixAp);
	}

#if DEBUG == 3
	printf("MatrixA");
	m_print(MatrixA);
	printf("MatrixCA");
	m_print(MatrixCA);
	printf("state");
	m_print(state);
#endif
}



















/**
 * Generates the signal on the CPU using the matrices generated earlier.
 * The signal is generated in chunks the size of synth.blocksize. The space for the resulting signal has been pre-
 * allocated earlier and is being filled by the filter.
 *
 * @param void
 * @return void
 */

void generateSignalCPU(float * output, String string, Synthesizer synth) {
	int i, j;
	Matrix state_tmp, output_chunk;

	Matrix *pointer_state_read, *pointer_state_write;

	m_new(&output_chunk, synth.blocksize,1);
	pointer_state_read = &state;
	pointer_state_write = &state_tmp;

	m_prepare_multiply(MatrixAp, state, &state_tmp);

	for(i = 0; i < synth.samples;) {
		m_multiply(MatrixCA, *pointer_state_read, &output_chunk);

		for(j = 0; j < synth.blocksize; j++) {
			output[i+j] = m_get(output_chunk,j,0)/128;
		}
		m_multiplyblockdiag(MatrixAp, *pointer_state_read, pointer_state_write, 2);
		m_swap(&pointer_state_read, &pointer_state_write);
		i = i + synth.blocksize;
	}
}



















/**
 * Generates the signal on the GPU using the matrices generated earlier.
 * The signal is generated in chunks the size of synth.blocksize. The space for the resulting signal has been pre-
 * allocated earlier and is being filled by the filter.
 *
 * @param void
 * @return void
 */

void generateSignalGPU(float * output, String string, Synthesizer synth) {
	int i, j;

	Matrix device_MatrixAp;
	Matrix device_MatrixCA;
	Matrix device_state_read, device_state_write;
	Matrix output_chunk_read, output_chunk_write;
	Matrix *pointer_output_chunk_read, *pointer_output_chunk_write;

	Matrix *pointer_device_state_read, *pointer_device_state_write;
	Matrix device_output_chunk_read, device_output_chunk_write;
	Matrix *pointer_device_output_chunk_read, *pointer_device_output_chunk_write;

	pointer_output_chunk_read = &output_chunk_read;
	pointer_output_chunk_write = &output_chunk_write;
	m_new(&output_chunk_read, synth.blocksize,1);
	m_new(&output_chunk_write, synth.blocksize,1);
	m_new(&device_output_chunk_read, synth.blocksize,1);
	m_new(&device_output_chunk_write, synth.blocksize,1);
	m_new(&device_MatrixCA, synth.blocksize, MatrixA.cols);
	m_new(&device_MatrixAp, MatrixA.rows, MatrixA.cols); // BLOCKDIAGMATRIX
	m_new(&device_state_read, 2 * synth.filters, 1);
	m_new(&device_state_write, 2 * synth.filters, 1);

	pointer_device_state_read = &device_state_read;
	pointer_device_state_write = &device_state_write;
	pointer_device_output_chunk_read = &device_output_chunk_read;
	pointer_device_output_chunk_write = &device_output_chunk_write;

	hipSetDevice(0);

	hipStream_t streams[3];

	hipEvent_t MatrixCA_start, MatrixCA_stop;
	hipEvent_t MatrixAp_start, MatrixAp_stop;
	hipEvent_t Memcpy_start, Memcpy_stop;

	hipEventCreate(&MatrixCA_start);
	hipEventCreate(&MatrixCA_stop);
	hipEventCreate(&MatrixAp_start);
	hipEventCreate(&MatrixAp_stop);
	hipEventCreate(&Memcpy_start);
	hipEventCreate(&Memcpy_stop);

	float MatrixCA_time, MatrixAp_time, Memcpy_time;


	for(int i = 0; i < 3; i++) {
		hipStreamCreate(& streams[i]);
	}

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixAp.elements, m_size(MatrixAp)));
	CUDA_SAFE_CALL(hipMemcpy(device_MatrixAp.elements, MatrixAp.elements, m_size(MatrixAp), hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixCA.elements, m_size(MatrixCA)));
	CUDA_SAFE_CALL(hipMemcpy(device_MatrixCA.elements, MatrixCA.elements, m_size(MatrixCA), hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_output_chunk_read.elements, m_size(output_chunk_read)));
	CUDA_SAFE_CALL(hipMemcpy(device_output_chunk_read.elements, output_chunk_read.elements, m_size(output_chunk_read), hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_output_chunk_write.elements, m_size(output_chunk_write)));
	CUDA_SAFE_CALL(hipMemcpy(device_output_chunk_write.elements, output_chunk_write.elements, m_size(output_chunk_write), hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_state_read.elements, m_size(state)));
	CUDA_SAFE_CALL(hipMemcpy(device_state_read.elements, state.elements, m_size(state), hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_state_write.elements, m_size(state)));
	CUDA_SAFE_CALL(hipMemcpy(device_state_write.elements, state.elements, m_size(state), hipMemcpyHostToDevice));

	dim3 dimBlockCA(1, synth.blocksize/10); // @TODO Optimierungspotential
	dim3 dimGridCA(state.cols / dimBlockCA.x, MatrixCA.rows / dimBlockCA.y);

	dim3 dimBlockA(1, synth.blocksize/10); // @TODO Optimierungspotential
	dim3 dimGridA(state.cols / dimBlockA.x, MatrixAp.rows / dimBlockA.y);

	hipEventRecord(MatrixCA_start, streams[0]);
	MatrixMultiplyKernel<<<dimGridCA, dimBlockCA, 1, streams[0]>>>(device_MatrixCA, *pointer_device_state_read, *pointer_device_output_chunk_write);
	hipEventRecord(MatrixCA_stop, streams[0]);

	hipEventRecord(MatrixAp_start, streams[1]);
	MatrixMultiplyKernel<<<dimGridA, dimBlockA, 1, streams[1]>>>(device_MatrixAp, *pointer_device_state_read, *pointer_device_state_write);
	hipEventRecord(MatrixAp_stop, streams[1]);

	for(i = 0; i < synth.samples;) {
		/*
	       	 * CUDA IMPLEMENTATION
		 */

		hipDeviceSynchronize();

		hipEventElapsedTime(&MatrixCA_time, MatrixCA_start, MatrixCA_stop);
		hipEventElapsedTime(&MatrixAp_time, MatrixAp_start, MatrixAp_stop);
		hipEventElapsedTime(&Memcpy_time, Memcpy_start, Memcpy_stop);

#if DEBUG == 10
		if(i == 5*synth.blocksize) {
			printf("MatrixCA: %d\n", MatrixCA_time);
			printf("MatrixAp: %d\n", MatrixAp_time);
			printf("  Memcpy: %d\n", Memcpy_time);
		}
#endif

		m_swap(&pointer_device_state_read, &pointer_device_state_write);
		m_swap(&pointer_device_output_chunk_read, &pointer_device_output_chunk_write);
		m_swap(&pointer_output_chunk_read, &pointer_output_chunk_write);

		hipEventRecord(MatrixCA_start, streams[0]);
		MatrixMultiplyKernel<<<dimGridCA, dimBlockCA, 1, streams[0]>>>(device_MatrixCA, *pointer_device_state_read, *pointer_device_output_chunk_write);
		hipEventRecord(MatrixCA_stop, streams[0]);

		hipEventRecord(MatrixAp_start, streams[1]);
		BlockDiagMatrixMultiplyKernel<<<dimGridA, dimBlockA, 1, streams[1]>>>(device_MatrixAp, *pointer_device_state_read, *pointer_device_state_write, 2);
		hipEventRecord(MatrixAp_stop, streams[1]);

		hipEventRecord(Memcpy_start, streams[2]);
		hipMemcpyAsync(pointer_output_chunk_write->elements, pointer_device_output_chunk_read->elements, m_size(output_chunk_write), hipMemcpyDeviceToHost, streams[2]);
		hipEventRecord(Memcpy_stop, streams[2]);

		for(j = 0; j < synth.blocksize; j++) {
			output[i+j] = m_get(*pointer_output_chunk_read,j,0)/128;
		}

		i = i + synth.blocksize;
	}
}




















/**
 * The values from output are passed on to the sndfile library and
 * written to the file `filter.wav`.
 *
 * @param void
 * @return void
 */

void writeFile(const char * filename, float* input, int samples, int samplerate) {
	SF_INFO info;
	info.format = SF_FORMAT_WAV | SF_FORMAT_PCM_16;
	info.channels = 1;
	info.samplerate = samplerate;

	SNDFILE *outfile = sf_open(filename, SFM_WRITE, &info);
	assert(outfile);
	sf_writef_float(outfile, input, samples);
}
