#include "hip/hip_runtime.h"
#include "filter.h"

String string;
Synthesizer synth;
Matrix MatrixC, MatrixA, state;
Matrix MatrixAp, MatrixCA;
Matrix device_MatrixAp, device_MatrixCA;
Timer turnaround, overall;
int xmloutput;

/**
 * Wrapper for the methods required in the filter, just calls them in the correct order
 *
 * @param int length
 * @return int 0
 */

int filter() {
	time_start(&overall);
	time_start(&turnaround);
	initializeCoefficients();

	float * output = (float *) malloc(sizeof(float) * synth.samples);
	createMatrices();

	if(settings.mode == 0) {
		createBlockprocessingMatrices();
		generateSignalCPU(output, string, synth);
	}
	else {
		initializeGPU();
		createBlockprocessingMatricesGPU();
		generateSignalGPU(output, string, synth);
	}

	writeFile("filter.wav", output, synth.samples, synth.T);
	time_stop(&overall);
	time_print(&overall, "overall");
	return 0;
}











/**
 * Initializes coefficients for a standard Nylon-b-String. The values are, with exception for
 * length (l) are hard-coded according to the values in \cite{rabenstein03}.
 *
 * @param float length
 * @return void
 */

void initializeCoefficients() {
	// Saiten-Koeffizienten
	string.l = settings.length;
	string.Ts = 60.97;
	string.rho = 1140;
	string.A = 0.5188e-6;
	string.E = 5.4e9;
	string.I = 0.171e-12;
	string.d1 = 8e-5;
	string.d3 = -1.4e-5;

	// Abtastpunkt
	string.xa = 0.1;

	// Abtastrate und Samplelänge
	synth.T = 44100;
	synth.seconds = 10;
	synth.samples = settings.samples;
	synth.filters = settings.filters;
	synth.blocksize = settings.chunksize;

	assert(synth.samples % synth.blocksize == 0);
}















/**
 * Creates the required matrices by calculating the required number of poles using the equations
 * to be found in \cite{rabenstein03}. The matrices generated are not yet in blockprocessing form.
 *
 * @param void
 * @return void
 */

void createMatrices() {
	int i, mu;
	double gamma, sigma;
	double omega;
	double a, b, c1, c0;

	m_new(&MatrixC, 1, 2 * synth.filters);
	m_new(&MatrixA, 2 * synth.filters, 2 * synth.filters); //BLOCKMATRIX
	m_new(&state, 2 * synth.filters, 1);

	Timer timer;
	time_start(&timer);

	for(i = 0; i < synth.filters; i++) {
		mu = i+1;
		gamma = mu * ( M_PI / string.l );
		sigma = (1 / (2 * string.rho * string.A) ) * (string.d3 * pow(gamma,2) - string.d1);
		omega = sqrt(
				  (
					(
						(string.E * string.I)/(string.rho * string.A)
					      - pow(string.d3, 2)/pow(2 * string.rho * string.A, 2)
					) * pow(gamma, 4)
				  )
				+ (	(
						(string.Ts)/(string.rho * string.A) 
					      + (string.d1 + string.d3)/(2*pow(string.rho*string.A,2))
					) * pow(gamma, 2) )
				+ (
					pow((string.d1)/(2 * string.rho * string.A), 2)
				  )
			);

		a = sin(mu * M_PI * string.xa / string.l)/128;

		b = synth.T * sin(omega * 1 / synth.T) / (omega * 1 / synth.T);
		c1 = -2 * exp(sigma * 1 / synth.T) * cos(omega * 1 / synth.T);
		c0 = exp( 2 * sigma * 1 / synth.T);

		noop(&b);

		m_set(&MatrixC, 0, 2*i  , 0);
		m_set(&MatrixC, 0, 2*i+1, a);

		m_set(&MatrixA, 2*i  , 2*i  , 0);
		m_set(&MatrixA, 2*i  , 2*i+1, -c0);
		m_set(&MatrixA, 2*i+1, 2*i  , 1);
		m_set(&MatrixA, 2*i+1, 2*i+1, -c1);

		m_set(&state, 2*i  , 0, 0);
		m_set(&state ,2*i+1, 0, 1);

	}

	time_stop(&timer);
	time_print(&timer, "CreateM");
}
















/**
 * Generates matrices suitable for blockprocessing
 *
 * @param int blocksize
 * @return void
 */ 

void createBlockprocessingMatricesGPU() {
	int i;
	Timer timer;

	Matrix device_MatrixCA_line_read, device_MatrixCA_line_write;
	Matrix device_MatrixAp_read, device_MatrixAp_write;
	Matrix device_MatrixC, device_MatrixA;

	Matrix *pointer_device_MatrixCA_line_read, *pointer_device_MatrixCA_line_write;
	Matrix *pointer_device_MatrixAp_read, *pointer_device_MatrixAp_write;

	hipStream_t streams[3];

	pointer_device_MatrixAp_write = &device_MatrixAp_write;
	pointer_device_MatrixAp_read = &device_MatrixAp_read;
	pointer_device_MatrixCA_line_write = &device_MatrixCA_line_write;
	pointer_device_MatrixCA_line_read = &device_MatrixCA_line_read;

	time_start(&timer);

	for(int i = 0; i < 3; i++) {
		hipStreamCreate(& streams[i]);
	}

	m_new(&MatrixCA, synth.blocksize, MatrixA.cols);
	m_new(&MatrixAp, MatrixA.rows, MatrixA.cols); // BLOCKDIAGMATRIX
	m_new(&device_MatrixA, MatrixA.rows, MatrixA.cols); // BLOCKDIAGMATRIX
	m_new(&device_MatrixCA, synth.blocksize, MatrixA.cols);
	m_new(&device_MatrixAp, MatrixA.rows, MatrixA.cols); // BLOCKDIAGMATRIX
	m_new(&device_MatrixAp_write, MatrixA.rows, MatrixA.cols); // BLOCKDIAGMATRIX
	m_new(&device_MatrixAp_read, MatrixA.rows, MatrixA.cols); // BLOCKDIAGMATRIX

	m_new(&device_MatrixC, MatrixC.rows, MatrixC.cols); // BLOCKDIAGMATRIX

	m_prepare_multiply(&MatrixC, &MatrixAp, &device_MatrixCA_line_read);
	m_prepare_multiply(&MatrixC, &MatrixAp, &device_MatrixCA_line_write);
	m_prepare_multiply(&MatrixAp, &MatrixA, &device_MatrixAp_read);
	m_prepare_multiply(&MatrixAp, &MatrixA, &device_MatrixAp_write);

	m_identity(&MatrixAp);

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixAp.elements, m_size(&MatrixAp)));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixAp_read.elements, m_size(&MatrixAp)));
	CUDA_SAFE_CALL(hipMemcpyAsync(device_MatrixAp_read.elements, MatrixAp.elements, m_size(&MatrixAp), hipMemcpyHostToDevice, streams[0]));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixAp_write.elements, m_size(&MatrixAp)));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixA.elements, m_size(&MatrixA)));
	CUDA_SAFE_CALL(hipMemcpyAsync(device_MatrixA.elements, MatrixA.elements, m_size(&MatrixA), hipMemcpyHostToDevice, streams[1]));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixCA.elements, m_size(&MatrixCA)));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixC.elements, m_size(&MatrixC)));
	CUDA_SAFE_CALL(hipMemcpyAsync(device_MatrixC.elements, MatrixC.elements, m_size(&MatrixC), hipMemcpyHostToDevice, streams[2]));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixCA_line_read.elements, m_size(&device_MatrixCA_line_read)));
	CUDA_SAFE_CALL(hipMalloc((void**) &device_MatrixCA_line_write.elements, m_size(&device_MatrixCA_line_write)));

	dim3 dimBlockCA(1, 1); // @TODO Optimierungspotential
	dim3 dimGridCA(MatrixAp.cols / dimBlockCA.x, MatrixC.rows / dimBlockCA.y);

	dim3 dimBlockA(1, 1); // @TODO Optimierungspotential; groessere Werte sind kleinere Gridsize
	dim3 dimGridA(MatrixA.cols / dimBlockA.x, MatrixAp.rows / dimBlockA.y);

	hipDeviceSynchronize();

	for(i = 1; i <= synth.blocksize; i++) {
		
		MatrixMultiplyKernel<<<dimGridCA, dimBlockCA, 1, streams[0]>>>(device_MatrixC, *pointer_device_MatrixAp_read, *pointer_device_MatrixCA_line_write);
		// m_multiply(&MatrixC, pointer_MatrixAp, &MatrixCA_line);

		BlockDiagMatrixMultiplyKernel<<<dimGridA, dimBlockA, 1, streams[2]>>>(*pointer_device_MatrixAp_read, device_MatrixA, *pointer_device_MatrixAp_write, 2);
		// m_multiplyblockdiag(pointer_MatrixAp, &MatrixA, pointer_MatrixAp_tmp, 2);

		hipDeviceSynchronize();

		m_swap(&pointer_device_MatrixAp_write, &pointer_device_MatrixAp_read);
		m_swap(&pointer_device_MatrixCA_line_write, &pointer_device_MatrixCA_line_read);

		CUDA_SAFE_CALL(hipMemcpyAsync(&device_MatrixCA.elements[(i-1) * MatrixCA.cols], pointer_device_MatrixCA_line_read->elements, m_size(&device_MatrixCA_line_read), hipMemcpyDeviceToDevice, streams[1]));
	}

	hipDeviceSynchronize();

	CUDA_SAFE_CALL(hipMemcpyAsync(device_MatrixAp.elements, pointer_device_MatrixAp_read->elements, m_size(&MatrixAp), hipMemcpyDeviceToDevice, streams[0]));


	time_stop(&timer);
	time_print(&timer, "BlockprocM");
}
















/**
 * Generates matrices suitable for blockprocessing
 *
 * @param int blocksize
 * @return void
 */ 

void createBlockprocessingMatrices() {
	int i, j;
	Matrix MatrixCA_line, MatrixAp_tmp;
	Matrix *pointer_MatrixAp, *pointer_MatrixAp_tmp;

	pointer_MatrixAp = &MatrixAp;
	Timer timer;

	pointer_MatrixAp_tmp = &MatrixAp_tmp;

	m_new(&MatrixCA, synth.blocksize, MatrixA.cols);
	m_new(&MatrixAp, MatrixA.rows, MatrixA.cols); // BLOCKDIAGMATRIX
	m_identity(&MatrixAp);

	m_prepare_multiply(&MatrixC, &MatrixAp, &MatrixCA_line);
	m_prepare_multiply(&MatrixAp, &MatrixA, &MatrixAp_tmp);

	for(i = 1; i <= synth.blocksize; i++) {
		time_start(&timer);
		m_multiply(&MatrixC, pointer_MatrixAp, &MatrixCA_line);
		time_stop(&timer);
		time_print(&timer, "BlockprocM");

		for(j = 0; j < MatrixCA_line.cols; j++) {
			m_set(&MatrixCA, i-1, j, m_get(&MatrixCA_line, 0, j));
		}

		m_multiplyblockdiag(pointer_MatrixAp, &MatrixA, pointer_MatrixAp_tmp, 2);

		m_swap(&pointer_MatrixAp_tmp, &pointer_MatrixAp);
	}

	time_stop(&timer);
	time_print(&timer, "BlockprocM");
}



















/**
 * Initializes the GPU
 *
 * @param void
 * @return void
 */

void initializeGPU() {
	hipSetDevice(0);
}



















/**
 * Generates the signal on the CPU using the matrices generated earlier.
 * The signal is generated in chunks the size of synth.blocksize. The space for the resulting signal has been pre-
 * allocated earlier and is being filled by the filter.
 *
 * @param void
 * @return void
 */

void generateSignalCPU(float * output, String string, Synthesizer synth) {
	int i;
	Matrix state_tmp, output_chunk;
	Matrix *pointer_state_read, *pointer_state_write;
	Timer roundtrip;

	m_new(&output_chunk, synth.blocksize,1);
	pointer_state_read = &state;
	pointer_state_write = &state_tmp;

	m_prepare_multiply(&MatrixAp, &state, &state_tmp);

	for(i = 0; i < synth.samples;) {
		time_start(&roundtrip);
		m_multiply(&MatrixCA, pointer_state_read, &output_chunk);

		memcpy(&output[i], output_chunk.elements, sizeof(float) * synth.blocksize);

		m_multiplyblockdiag(&MatrixAp, pointer_state_read, pointer_state_write, 2);
		m_swap(&pointer_state_read, &pointer_state_write);
		if(i == 0) {
			time_stop(&turnaround);
			time_print(&turnaround, "turnaround");
		}
		time_stop(&roundtrip);

		if(i == 5*synth.blocksize) {
			time_print(&roundtrip, "roundtrip");
		}

		i = i + synth.blocksize;
	}
}



















/**
 * Generates the signal on the GPU using the matrices generated earlier.
 * The signal is generated in chunks the size of synth.blocksize. The space for the resulting signal has been pre-
 * allocated earlier and is being filled by the filter.
 *
 * @param void
 * @return void
 */

void generateSignalGPU(float * output, String string, Synthesizer synth) {
	int i;

	Matrix device_state_read, device_state_write;
	Matrix output_chunk_read, output_chunk_write;
	Matrix *pointer_output_chunk_read, *pointer_output_chunk_write;

	Matrix *pointer_device_state_read, *pointer_device_state_write;
	Matrix device_output_chunk_read, device_output_chunk_write;
	Matrix *pointer_device_output_chunk_read, *pointer_device_output_chunk_write;

	Timer roundtrip;

	pointer_output_chunk_read = &output_chunk_read;
	pointer_output_chunk_write = &output_chunk_write;
	m_new(&output_chunk_read, synth.blocksize,1);
	m_new(&output_chunk_write, synth.blocksize,1);
	m_new(&device_output_chunk_read, synth.blocksize,1);
	m_new(&device_output_chunk_write, synth.blocksize,1);
	m_new(&device_state_read, 2 * synth.filters, 1);
	m_new(&device_state_write, 2 * synth.filters, 1);

	pointer_device_state_read = &device_state_read;
	pointer_device_state_write = &device_state_write;
	pointer_device_output_chunk_read = &device_output_chunk_read;
	pointer_device_output_chunk_write = &device_output_chunk_write;

	hipStream_t streams[3];

	hipEvent_t MatrixCA_start, MatrixCA_stop;
	hipEvent_t MatrixAp_start, MatrixAp_stop;
	hipEvent_t Memcpy_start, Memcpy_stop;

	hipEventCreate(&MatrixCA_start);
	hipEventCreate(&MatrixCA_stop);
	hipEventCreate(&MatrixAp_start);
	hipEventCreate(&MatrixAp_stop);
	hipEventCreate(&Memcpy_start);
	hipEventCreate(&Memcpy_stop);

	float MatrixCA_time, MatrixAp_time, Memcpy_time;


	for(int i = 0; i < 3; i++) {
		hipStreamCreate(& streams[i]);
	}

	CUDA_SAFE_CALL(hipMalloc((void**) &device_state_read.elements, m_size(&state)));
	CUDA_SAFE_CALL(hipMemcpyAsync(device_state_read.elements, state.elements, m_size(&state), hipMemcpyHostToDevice, streams[2]));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_state_write.elements, m_size(&state)));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_output_chunk_read.elements, m_size(&output_chunk_read)));

	CUDA_SAFE_CALL(hipMalloc((void**) &device_output_chunk_write.elements, m_size(&output_chunk_write)));

	dim3 dimBlockCA(1, 1); // @TODO Optimierungspotential
	dim3 dimGridCA(state.cols / dimBlockCA.x, MatrixCA.rows / dimBlockCA.y);

	dim3 dimBlockA(1, 1); // @TODO Optimierungspotential; groessere Werte sind kleinere Gridsize
	dim3 dimGridA(state.cols / dimBlockA.x, MatrixAp.rows / dimBlockA.y);

	hipDeviceSynchronize();

	for(i = -synth.blocksize; i < synth.samples;) {
		/*
	       	 * CUDA IMPLEMENTATION
		 */

		hipDeviceSynchronize();
		time_stop(&roundtrip);

		if(i == 5*synth.blocksize) {
			time_print(&roundtrip, "roundtrip");
		}

		time_start(&roundtrip);

		hipEventElapsedTime(&MatrixCA_time, MatrixCA_start, MatrixCA_stop);
		hipEventElapsedTime(&MatrixAp_time, MatrixAp_start, MatrixAp_stop);
		hipEventElapsedTime(&Memcpy_time, Memcpy_start, Memcpy_stop);

		if(i >= 0) {
			m_swap(&pointer_device_state_read, &pointer_device_state_write);
			m_swap(&pointer_device_output_chunk_read, &pointer_device_output_chunk_write);
			m_swap(&pointer_output_chunk_read, &pointer_output_chunk_write);
		}
		hipEventRecord(MatrixCA_start, streams[0]);
		MatrixMultiplyKernel<<<dimGridCA, dimBlockCA, 1, streams[0]>>>(device_MatrixCA, *pointer_device_state_read, *pointer_device_output_chunk_write);
		hipEventRecord(MatrixCA_stop, streams[0]);

		hipEventRecord(MatrixAp_start, streams[1]);
		BlockDiagMatrixMultiplyKernel<<<dimGridA, dimBlockA, 1, streams[1]>>>(device_MatrixAp, *pointer_device_state_read, *pointer_device_state_write, 2);
		hipEventRecord(MatrixAp_stop, streams[1]);

		if(i >= 0) {
			hipEventRecord(Memcpy_start, streams[2]);
			hipMemcpyAsync(pointer_output_chunk_write->elements, pointer_device_output_chunk_read->elements, m_size(&output_chunk_write), hipMemcpyDeviceToHost, streams[2]);
			hipEventRecord(Memcpy_stop, streams[2]);

			memcpy(&output[i], pointer_output_chunk_read->elements, sizeof(float) * synth.blocksize);

			if(i == 0) {
				time_stop(&turnaround);
				time_print(&turnaround, "turnaround");
			}
		}

		i = i + synth.blocksize;
	}
}




















/**
 * The values from output are passed on to the sndfile library and
 * written to the file `filter.wav`.
 *
 * @param void
 * @return void
 */

void writeFile(const char * filename, float* input, int samples, int samplerate) {
	SF_INFO info;
	info.format = SF_FORMAT_WAV | SF_FORMAT_PCM_16;
	info.channels = 1;
	info.samplerate = samplerate;

	SNDFILE *outfile = sf_open(filename, SFM_WRITE, &info);
	assert(outfile);
	sf_writef_float(outfile, input, samples);
}
