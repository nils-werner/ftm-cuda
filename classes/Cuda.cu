#include "Cuda.h"

Cuda::copyToDevice() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_A.elements, size));
	CUDA_SAFE_CALL(hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice));
	*/
}

Cuda::copyToHost() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_B.elements, size));
	CUDA_SAFE_CALL(hipMemcpy(d_B.elements, B.elements, size, hipMemcpyDeviceToHost));
	*/
}

Cuda::malloc() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_C.elements, size));
	*/
}

Cuda::free() {
	/*
	hipFree(d_A.elements);
	*/
}

Cuda::invoke() {
	/*
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	CUT_CHECK_ERROR("Kernel execution failed\n");
	*/
}
