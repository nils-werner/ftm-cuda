#include "Cuda.h"

void Cuda::copyToDevice() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_A.elements, size));
	CUDA_SAFE_CALL(hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice));
	*/
}

void Cuda::copyToHost() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_B.elements, size));
	CUDA_SAFE_CALL(hipMemcpy(d_B.elements, B.elements, size, hipMemcpyDeviceToHost));
	*/
}

void Cuda::malloc() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_C.elements, size));
	*/
}

void Cuda::free() {
	/*
	hipFree(d_A.elements);
	*/
}

void Cuda::invoke() {
	/*
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	CUT_CHECK_ERROR("Kernel execution failed\n");
	*/
}
