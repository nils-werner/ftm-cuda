#include "hip/hip_runtime.h"
#include "Cuda.h"

__global__ void CudaClassKernel() {
};

void Cuda::copyToDevice(void* hostData, void* deviceData, size_t size) {
	CUDA_SAFE_CALL(hipMalloc((void**)&deviceData, size));
	CUDA_SAFE_CALL(hipMemcpy(deviceData, hostData, size, hipMemcpyHostToDevice));
}

void Cuda::copyToHost(void* deviceData, void* hostData, size_t size) {
	CUDA_SAFE_CALL(hipMalloc((void**)&deviceData, size));
	CUDA_SAFE_CALL(hipMemcpy(deviceData, hostData, size, hipMemcpyDeviceToHost));
}

void Cuda::malloc(void* devicePtr, size_t size) {
	CUDA_SAFE_CALL(hipMalloc((void**)&devicePtr, size));
}

void Cuda::free(void* deviceData) {
	hipFree(deviceData);
}

void Cuda::invoke() {
	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid(1 / dimBlock.x, 1 / dimBlock.y);

	CudaClassKernel<<<dimGrid, dimBlock>>>();
	CUT_CHECK_ERROR("Kernel execution failed\n");
}
