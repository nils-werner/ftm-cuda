#include "Cuda.h"

void Cuda::copyToDevice() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_A.elements, size));
	CUDA_SAFE_CALL(hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice));
	*/
}

void Cuda::copyToHost() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_B.elements, size));
	CUDA_SAFE_CALL(hipMemcpy(d_B.elements, B.elements, size, hipMemcpyDeviceToHost));
	*/
}

void Cuda::malloc() {
	/*
	CUDA_SAFE_CALL(hipMalloc((void**)&d_C.elements, size));
	*/
}

void Cuda::free() {
	/*
	hipFree(d_A.elements);
	*/
}

void Cuda::invoke() {
	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid(1 / dimBlock.x, 1 / dimBlock.y);

	CudaClassKernel<<<dimGrid, dimBlock>>>();
	CUT_CHECK_ERROR("Kernel execution failed\n");
}
