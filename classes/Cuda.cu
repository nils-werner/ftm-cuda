#include "hip/hip_runtime.h"
#include "Cuda.h"

__global__ void CudaClassKernel() {
};

Cuda::Cuda() {
	setDevice(0);
}

void Cuda::setDevice(int deviceID) {
	assert(!hipSetDevice(deviceID));
}

void Cuda::copyToDevice(void* hostData, void* deviceData, size_t size) {
	//CUDA_SAFE_CALL(hipMalloc((void**)&deviceData, size));
	mallocOnDevice(&deviceData, size);
	CUDA_SAFE_CALL(hipMemcpy(deviceData, hostData, size, hipMemcpyHostToDevice));
}

void Cuda::copyToHost(void* deviceData, void* hostData, size_t size) {
	CUDA_SAFE_CALL(hipMemcpy(deviceData, hostData, size, hipMemcpyDeviceToHost));
}

void Cuda::mallocOnDevice(void** devicePtr, size_t size) {
	CUDA_SAFE_CALL(hipMalloc((void**) &devicePtr, size));
}

void Cuda::free(void* deviceData) {
	hipFree(deviceData);
}

void Cuda::invoke() {
	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid(1 / dimBlock.x, 1 / dimBlock.y);

	CudaClassKernel<<<dimGrid, dimBlock>>>();
	CUT_CHECK_ERROR("Kernel execution failed\n");
}
